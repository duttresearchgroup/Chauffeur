#include "hip/hip_runtime.h"
using namespace std;

#include "detector.h"
#include "polifitgsl.h"
#include "opencv2/gpu/gpu.hpp"
//#include "thrust/device_vector.h"
//#include "thrust/host_vector.h"

#include <libconfig.h++>

using namespace cv;

//-----CLASS METHOD DECLARATIONS-----//

int polynomial(const double *params, int degree, double x);
void thresh(gpu::GpuMat &src, gpu::GpuMat &dst);

//-----CLASS METHODS-----//

Detector::Detector(string config_path)
{
    libconfig::Config cfg;
    try
    {
        cfg.readFile(config_path.c_str());
        threshold = cfg.lookup("detector.threshold");
        row_step = cfg.lookup("detector.row_step");
        col_step = cfg.lookup("detector.col_step");
        l_start = cfg.lookup("detector.start.left");
        r_start = cfg.lookup("detector.start.right");
        
        string path = cfg.lookup("video.file");
        VideoCapture cap(path);
        Mat frame;
        cap >> frame;
        
        matrix_transform_birdseye = getTransformMatrix(frame);
        matrix_transform_fiperson = getTransformMatrix(frame, true);
        cap.release();
    }
    catch(...)
    {
        cerr << "Invalid config file" << endl;
    }
}

/**
 * Get lanes
 * @param img processed (thresholded and warped to birdseye perpective) frame from video
 * @param lane Detected lane
 */
void Detector::getLanes(const Mat &img, Lane &lane)
{                
    gpu::GpuMat gpu_img;
    gpu::GpuMat th;
    gpu::GpuMat warped;
    Mat dst;
    
    gpu_img.upload(img);
    thresh(gpu_img, th);
    gpu::warpPerspective(th, warped, matrix_transform_birdseye, Size(img.cols, img.rows));
    warped.download(dst);
    
    int width = dst.cols;
    int height = dst.rows;
    
    int left = width * l_start / 100;
    int right = width * r_start / 100;
    
    vector<double> lx;
    vector<double> rx;
    vector<double> ly;
    vector<double> ry;
    
    //Loop through frame rows at row_step
    for (int i = height-1; i >= 0; i-=row_step)
    {
        //Loop through left side
        lx.push_back(left);
        ly.push_back(i);
        for (int j = 0; j <= threshold; j+=col_step)
        {
            if (dst.at<uchar>(i, left+j) == 255) 
            {
                lx.back() = left+j;
                left += j;
                break;
            }
            
            if (dst.at<uchar>(i, left-j) == 255) 
            {
                lx.back() = left-j;
                left -= j;
                break;
            }
        }
        
        //Loop through right side
        rx.push_back(right);
        ry.push_back(i);
        for (int j = 0; j <= threshold; j+=col_step)
        {
            if (dst.at<uchar>(i, right-j) == 255) 
            {
                rx.back() = right-j;
                right -= j;
                break;
            }
            
            if (dst.at<uchar>(i, right+j) == 255) 
            {
                rx.back() = right+j;
                right += j;
                break;
            }
        }
         
    }
    
    vector<double> l_new(lane.getN(), 0.0);
    vector<double> r_new(lane.getN(), 0.0);
    polynomialfit(lx.size(), lane.getN(), &ly[0], &lx[0], &l_new[0]);
    polynomialfit(rx.size(), lane.getN(), &ry[0], &rx[0], &r_new[0]);
    
    lane.update(l_new, r_new);
}

/**
 * Draws lane on an image
 * @param img Image on which to draw lane
 * @param lane Lane to draw
 * @param m perspective transform matrix for lane
 */
void Detector::drawLane(Mat &img, Lane &lane)
{
    Mat blank(img.size(), img.type(), Scalar(0, 0, 0));
    for (int i = 0; i < img.rows; i++)
    {
        circle(blank, Point(polynomial(&(lane.getParams())[0], lane.getN(), i), i), 3, Scalar(150, 0, 0), 3);
    }
    warpPerspective(blank, blank, matrix_transform_fiperson, Size(img.cols, img.rows));
    for (int i = 0; i < img.rows; i+=2)
    {
        for (int j = 0; j < img.cols; j+=2)
        {
            if (blank.at<Vec3b>(i, j)[0] == 150)
                circle(img, Point(j, i), 1, Scalar(150, 0, 0), 1);
        }
    }
}



/**
 * Gets the perpective transform matrix for warping an image to birdseye perspective
 * @param img Image to generate matrix for
 * @param undo If undo is true, return the matrix for transforming from birdseye to first-person perspective
 * @return perspective transform matrix
 */
Mat Detector::getTransformMatrix(Mat img, bool undo)
{
    int width = img.cols;
    int height = img.rows;
    vector<Point2f> src = {Point2f(width*0.44,height*0.20), Point2f(width*0.56,height*0.20), Point2f(width*1.00,height*0.85), Point2f(width*0.00,height*0.85)};
    vector<Point2f> dst = {Point2f(width*0.20,height*0.00), Point2f(width*0.80,height*0.00), Point2f(width*0.80,height*1.00), Point2f(width*0.20,height*1.00)};
    
    Mat m;
    if (undo) m = getPerspectiveTransform(&dst[0], &src[0]);
    else m = getPerspectiveTransform(&src[0], &dst[0]);
    return m;
}


//-----NON CLASS METHODS-----//


void thresh(gpu::GpuMat &src, gpu::GpuMat &dst)
{
    gpu::cvtColor(src, dst, CV_BGR2GRAY);
    
    gpu::GaussianBlur(dst, dst, Size( 7, 7 ), 1.5, 1.5 );
    gpu::threshold(dst, dst, 185, 255, THRESH_BINARY);
}

/**
 * Evaluates a polynomial expression
 * @param params Array of polynomial coefficients
 * @param degree Degree of polynomial (size of params)
 * @param x Polynomial input
 * @return Evaluated expression.
 */
int polynomial(const double *params, int degree, double x)
{
    double val = 0;
    for (int i = 0; i < degree; i++)
    {
        val += params[i] * pow(x, i);
    }
    return (int)val;
}	
