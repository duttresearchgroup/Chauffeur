#include "hip/hip_runtime.h"
/**
 * Detect.cu
 * Provides functions for lane detection
 * 
 * @author Oscar Morales Ponce
 * @author Jared Coleman
 * @version 1.0 12/02/17
 */

using namespace std;

#include <fstream>
#include <string>
#include <cmath>
#include <libconfig.h++>

#include "opencv2/opencv.hpp"
#include "opencv2/gpu/gpu.hpp"
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"

#include "polifitgsl.h"

using namespace cv;


/**
 * Gets the perpective transform matrix for warping an image to birdseye perspective
 * @param img Image to generate matrix for
 * @param undo If undo is true, return the matrix for transforming from birdseye to first-person perspective
 * @return perspective transform matrix
 */
Mat getTransformMatrix(Mat img, bool undo=false)
{
    int width = img.cols;
    int height = img.rows;
    vector<Point2f> src = {Point2f(width*0.44,height*0.20), Point2f(width*0.56,height*0.20), Point2f(width*1.00,height*0.85), Point2f(width*0.00,height*0.85)};
    vector<Point2f> dst = {Point2f(width*0.20,height*0.00), Point2f(width*0.80,height*0.00), Point2f(width*0.80,height*1.00), Point2f(width*0.20,height*1.00)};
    
    Mat m;
    if (undo) m = getPerspectiveTransform(&dst[0], &src[0]);
    else m = getPerspectiveTransform(&src[0], &dst[0]);
    return m;
}

/**
 * Thresholds the image. Uses GPU acceleration
 * Process:
 *   1. Convert image to grayscale
 *   2. Blur the image to remove noise (gaussian)
 *   3. threshold image (binary)
 * @param src image to threshold (GpuMat)
 * @param dst destination for thresholded image (GpuMat)
 */
void thresh(gpu::GpuMat &src, gpu::GpuMat &dst)
{
    gpu::cvtColor(src, dst, CV_BGR2GRAY);
    
    gpu::GaussianBlur(dst, dst, Size( 7, 7 ), 1.5, 1.5 );
    gpu::threshold(dst, dst, 185, 255, THRESH_BINARY);
}

/**
 * Evaluates a polynomial expression
 * @param params Array of polynomial coefficients
 * @param degree Degree of polynomial (size of params)
 * @param x Polynomial input
 * @return Evaluated expression.
 */
int polynomial(const double *params, int degree, double x)
{
    double val = 0;
    for (int i = 0; i < degree; i++)
    {
        val += params[i] * pow(x, i);
    }
    return (int)val;
}	

/**
 * Get lanes
 * @param img processed (thresholded and warped to birdseye perpective) frame from video
 * @param lane Detected lane
 */
void getLanes(const Mat &img, Lane *lane)
{
    static int row_step = config.row_step;
    static int col_step = config.col_step;
    static int d = config.lane_start_threshold;
    
    int width = img.cols;
    int height = img.rows;
    
    int left = width * config.left_lane_start / 100;
    int right = width * config.right_lane_start / 100;
    
    vector<double> lx;
    vector<double> rx;
    vector<double> ly;
    vector<double> ry;
    
    //Loop through frame rows at row_step
    for (int i = height-1; i >= 0; i-=row_step)
    {
        //Loop through left side
        lx.push_back(left);
        ly.push_back(i);
        for (int j = left + d; j >= left - d; j-=col_step)
        {
            if (img.at<uchar>(i, j) == 255) 
            {
                lx.back() = j;
                left = j;
                break;
            }
        }
        
        //Loop through right side
        rx.push_back(right);
        ry.push_back(i);
        for (int j = right - d; j < right + d; j+=col_step)
        {
            if (img.at<uchar>(i, j) == 255) 
            {
                rx.back() = j;
                right = j;
                break;
            }
        }
         
    }
    
    vector<double> l_new(lane->getN(), 0.0);
    vector<double> r_new(lane->getN(), 0.0);
    polynomialfit(lx.size(), lane->getN(), &ly[0], &lx[0], &l_new[0]);
    polynomialfit(rx.size(), lane->getN(), &ry[0], &rx[0], &r_new[0]);
    
    lane->update(l_new, r_new);
}

/**
 * Draws lane on an image
 * @param img Image on which to draw lane
 * @param lane Lane to draw
 * @param m perspective transform matrix for lane
 */
void drawLane(Mat &img, const Lane *lane, Mat &m)
{
    Mat blank(img.size(), img.type(), Scalar(0, 0, 0));
    for (int i = 0; i < img.rows; i++)
    {
        circle(blank, Point(polynomial(&(lane->params)[0], lane->getN(), i), i), 3, Scalar(150, 0, 0), 3);
    }
    warpPerspective(blank, blank, m, Size(img.cols, img.rows));
    for (int i = 0; i < img.rows; i+=2)
    {
        for (int j = 0; j < img.cols; j+=2)
        {
            if (blank.at<Vec3b>(i, j)[0] == 150)
                circle(img, Point(j, i), 1, Scalar(150, 0, 0), 1);
        }
    }
}
 
void sendMessage(SerialCommunication *serial, int8_t angle)
{
    UARTCommand command1, command2;
    command1.speed = 0;
    command1.wheelOrientation = angle;
    command1.maxTime = 100;
    command1.orientation= 0;
    serial->sendCommand(&command1);
 
}

int main(int argc, char* argv[])
{
    if (argc < 2)
    {
        cout << "Usage: " << argv[0] << " <config file>" << endl;
        return 0;
    }
    
    Lane *lane;
    SerialCommunication *serial;
    VideoCapture *cap;
    Detector *detector;
    double *k;
    
    if (getConfig(argv[1], lane, serial, cap, detector, k))
    {
        cout << argv[1] << " is not a valid config file" << endl;
        return 0;
    }
    
    serial->run();
        
    if(!cap->isOpened()) return -1;
    
    //namedWindow("output", 1);
    Mat frame;
    *cap >> frame;
    Mat birdseye = getTransformMatrix(frame);
    Mat first_person = getTransformMatrix(frame, true);
    
    gpu::GpuMat src;
    gpu::GpuMat th;
    gpu::GpuMat dst;
    while(true)
    {
        //get frame from stream
        *cap >> frame;
        
        //copy of original frame for drawing
        Mat original = frame.clone();
        
        //preprocess img
        src.upload(frame);
        thresh(src, th);
        gpu::warpPerspective(th, dst, birdseye, Size(frame.cols, frame.rows));
        dst.download(frame);
    
        //Get lanes
        //getLanes(frame, lane);
        
        //draw lanes
        //drawLane(original, lane, first_person);
        
        //send lane
        sendMessage(serial, (int)(k * lane->curvature()));
        
        //-------------------------------------------------------//
        //imshow("output", original);
        if(waitKey(1) >= 0) break;
    }
}
